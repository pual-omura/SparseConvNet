
#include <hip/hip_runtime.h>
// Copyright 2016-present, Facebook, Inc.
// All rights reserved.
//
// This source code is licensed under the license found in the
// LICENSE file in the root directory of this source tree.

#ifndef TH_GENERIC_FILE_
#define TH_GENERIC_FILE_ "generic/GPU/SparseToDense.cu"
#else
#include "SparseToDense.h"

extern "C" void scn_DR_(SparseToDense_updateOutput)(THLongTensor *inputSize,
                                                    void **m,
                                                    THCTensor *input_features,
                                                    THCTensor *output_features,
                                                    THCITensor *rulesBuffer) {

  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m) {
    long sz[Dimension + 2];
    sz[0] = _m.inputSGs->size();
    sz[1] = input_features->size[1];
    for (int i = 0; i < Dimension; i++) {
      auto x = THLongTensor_data(inputSize)[i];
      sz[i + 2] = x;
    }
    THCTensor_(resizeNd)(state, output_features, Dimension + 2, sz, NULL);
    THCTensor_(zero)(state, output_features);
  }
  auto _rules = _m.getSparseToDenseRuleBook(inputSize, true);
  auto spatialVolume = _rules.size();
  uInt nPlanes = input_features->size[1];
  auto iF = THCTensor_(data)(state, input_features);
  auto oF = THCTensor_(data)(state, output_features);
  RULEBOOKITERATOR(
      SparseToDense_ForwardPass<real>(THCState_getCurrentStream(state), iF, oF,
                                      nPlanes, spatialVolume, rbB, nHotB);
      , oF++;) // todo check ++ or +=spatialVolume????zzz
}
extern "C" void scn_DR_(SparseToDense_updateGradInput)(
    THLongTensor *inputSize, void **m, THCTensor *input_features,
    THCTensor *d_input_features, THCTensor *d_output_features,
    THCITensor *rulesBuffer) {

  THCTensor_(resizeAs)(state, d_input_features, input_features);
  THCTensor_(zero)(state, d_input_features);

  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  auto _rules = _m.getSparseToDenseRuleBook(inputSize, true);
  auto spatialVolume = _rules.size();
  uInt nPlanes = d_input_features->size[1];
  auto diF = THCTensor_(data)(state, d_input_features);
  auto doF = THCTensor_(data)(state, d_output_features);
  RULEBOOKITERATOR(
      SparseToDense_BackwardPass<real>(THCState_getCurrentStream(state), diF,
                                       doF, nPlanes, spatialVolume, rbB, nHotB);
      , doF++;)
}
#endif
