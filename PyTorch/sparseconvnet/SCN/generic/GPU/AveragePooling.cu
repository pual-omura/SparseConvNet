
#include <hip/hip_runtime.h>
// Copyright 2016-present, Facebook, Inc.
// All rights reserved.
//
// This source code is licensed under the license found in the
// LICENSE file in the root directory of this source tree.

#ifndef TH_GENERIC_FILE_
#define TH_GENERIC_FILE_ "generic/GPU/AveragePooling.cu"
#else
#include "AveragePooling.h"
#include "RuleBookIterator.h"

extern "C" void scn_DR_(AveragePooling_updateOutput)(
    THLongTensor *inputSize, THLongTensor *outputSize, THLongTensor *poolSize,
    THLongTensor *poolStride, void **m, THCTensor *input_features,
    THCTensor *output_features, long nFeaturesToDrop, THCITensor *rulesBuffer) {

  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  uInt nPlanes = input_features->size[1] - nFeaturesToDrop;
  auto _rules =
      _m.getRuleBook(inputSize, outputSize, poolSize, poolStride, true);
  uInt nActive = _m.getNActive(outputSize);
  THCTensor_(resize2d)(state, output_features, nActive,
                       input_features->size[1] - nFeaturesToDrop);
  THCTensor_(zero)(state, output_features);

  auto iF = THCTensor_(data)(state, input_features) + nFeaturesToDrop;
  auto oF = THCTensor_(data)(state, output_features);
  RULEBOOKITERATOR(AveragePooling_ForwardPass<real>(
                       THCState_getCurrentStream(state), iF, oF, nPlanes,
                       input_features->size[1], output_features->size[1], rbB,
                       nHotB, _rules.size());
                   , )
}

extern "C" void scn_DR_(AveragePooling_updateGradInput)(
    THLongTensor *inputSize, THLongTensor *outputSize, THLongTensor *poolSize,
    THLongTensor *poolStride, void **m, THCTensor *input_features,
    THCTensor *d_input_features, THCTensor *d_output_features,
    long nFeaturesToDrop, THCITensor *rulesBuffer) {

  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  uInt nPlanes = input_features->size[1] - nFeaturesToDrop;
  auto _rules =
      _m.getRuleBook(inputSize, outputSize, poolSize, poolStride, true);
  uInt nActive = _m.getNActive(outputSize);
  THCTensor_(resizeAs)(state, d_input_features, input_features);
  THCTensor_(zero)(state, d_input_features);

  auto diF = THCTensor_(data)(state, d_input_features) + nFeaturesToDrop;
  auto doF = THCTensor_(data)(state, d_output_features);
  RULEBOOKITERATOR(AveragePooling_BackwardPass<real>(
                       THCState_getCurrentStream(state), diF, doF, nPlanes,
                       input_features->size[1], d_output_features->size[1], rbB,
                       nHotB, _rules.size());
                   , )
}
#endif
