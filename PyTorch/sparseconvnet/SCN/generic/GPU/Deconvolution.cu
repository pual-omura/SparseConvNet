
#include <hip/hip_runtime.h>
// Copyright 2016-present, Facebook, Inc.
// All rights reserved.
//
// This source code is licensed under the license found in the
// LICENSE file in the root directory of this source tree.

#ifndef TH_GENERIC_FILE_
#define TH_GENERIC_FILE_ "generic/GPU/Deconvolution.cu"
#else
#include "Convolution.h"
#include "Deconvolution.h"

#include <algorithm>

extern "C" double scn_DR_(Deconvolution_updateOutput)(
    THLongTensor *inputSize, THLongTensor *outputSize, THLongTensor *filterSize,
    THLongTensor *filterStride, void **m, THCTensor *input_features,
    THCTensor *output_features, THCTensor *weight, THCTensor *bias,
    long filterVolume, THCITensor *rulesBuffer) {
  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  auto _rules =
      _m.getRuleBook(outputSize, inputSize, filterSize, filterStride, true);
  uInt nActive = _m.getNActive(outputSize);
  THCTensor_(resize2d)(state, output_features, nActive, weight->size[1]);
  if (not bias)
    THCTensor_(zero)(state, output_features);

  auto iF = THCTensor_(data)(state, input_features);
  auto oF = THCTensor_(data)(state, output_features);
  auto ip = input_features->size[1];
  auto op = output_features->size[1];
  auto w = THCTensor_(data)(state, weight);
  double flops = 0;

  if (bias) {
    auto b = THCTensor_(data)(state, bias);
    for (uInt i = 0; i < op; i += 32) {
      uInt blockDim = min(32L, op - i);
      uInt gridDim = min(4096, nActive);
      Convolution_fp_bias
              << <gridDim, blockDim, 0, THCState_getCurrentStream(state)>>>
          (oF + i, b + i, op, op, nActive);
    }
  }
  uInt c = ip * op;
  RULEBOOKITERATOR(
      dDeconvolution_forward2<real>(iF, oF, w, rbB, nHotB, ip, ip, op, op,
                                    THCState_getCurrentStream(state));
      , w += c; flops += nHotB * c;)
  return flops;
}

extern "C" void scn_DR_(Deconvolution_backward)(
    THLongTensor *inputSize, THLongTensor *outputSize, THLongTensor *filterSize,
    THLongTensor *filterStride, void **m, THCTensor *input_features,
    THCTensor *d_input_features, THCTensor *d_output_features,
    THCTensor *weight, THCTensor *d_weight, THCTensor *d_bias,
    long filterVolume, THCITensor *rulesBuffer) {
  SCN_INITIALIZE_AND_REFERENCE(Metadata<Dimension>, m)
  auto _rules =
      _m.getRuleBook(outputSize, inputSize, filterSize, filterStride, true);
  uInt nActive = _m.getNActive(outputSize);
  THCTensor_(resizeAs)(state, d_input_features, input_features);
  THCTensor_(zero)(state, d_input_features);

  auto iF = THCTensor_(data)(state, input_features);
  auto diF = THCTensor_(data)(state, d_input_features);
  auto doF = THCTensor_(data)(state, d_output_features);
  auto ip = input_features->size[1];
  auto op = d_output_features->size[1];
  auto w = THCTensor_(data)(state, weight);
  auto dw = THCTensor_(data)(state, d_weight);
  uInt c = ip * op;
  RULEBOOKITERATOR(dDeconvolution_backward_dW2<real>(
                       iF, diF, doF, w, dw, rbB, nHotB, ip, ip, op, op,
                       THCState_getCurrentStream(state));

                   , w += c; dw += c;)

  if (d_bias) {
    auto db = THCTensor_(data)(state, d_bias);
    Convolution_bp_bias(doF, db, op, op, nActive,
                        THCState_getCurrentStream(state));
  }
}

#endif
